#include "hip/hip_runtime.h"

extern "C" {
#include "hybrid.cuh"
//#include "../graphio/graphio.h"
}

#include <stdio.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


#define cudaCheck(error) \
	if (error != hipSuccess) { \
    	printf("Fatal error: %s at %s:%d\n", \
    		hipGetErrorString(error), \
    		__FILE__, __LINE__); \
    	exit(1); \
 }


__global__  void cudaHSSSPKernel1 ( int *row_ptr, int *col_ind, int *weights,
                                    int *visited, int *distance, unsigned int *temp_distance,
                                    int nv, int ne )
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if ( visited[tid] != 0 )
    {
        visited[tid] = 0;

        for(int edge = row_ptr[tid]; edge < row_ptr[tid + 1]; edge++)
        {
            int nid = col_ind[edge];

            int w = weights[edge];
            int du = distance[tid];
            int newDist = du + w;

           if ((temp_distance[nid] > newDist) && (du != INT_MAX))
            {
            	atomicExch(&temp_distance[nid], newDist);
            }
        }
    }
}

  

__global__  void cudaHSSSPKernel2 ( int *row_ptr, int *col_ind, int *weights,
                                    int *visited, int *distance, unsigned int *temp_distance)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (distance[tid] > temp_distance[tid])
    {
        distance[tid] = temp_distance[tid];
        visited[tid] = 1;
    }

    temp_distance[tid] = distance[tid];
}


__global__  void cudaHSSSPKernel1MoreEdges ( int *row_ptr, int *col_ind, int *weights,
                                    int *visited, int *distance, unsigned int *temp_distance,
                                    int nv, int ne, int *min_edges )
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if ( visited[tid] != 0 )
    {
        visited[tid] = 0;

        if ((row_ptr[tid+1] - row_ptr[tid]) > *min_edges) // process only the nodes which has more than ... edges 
        {
	        for(int edge = row_ptr[tid]; edge < row_ptr[tid + 1]; edge++)
	        {
	            int nid = col_ind[edge];

	            int w = weights[edge];
	            int du = distance[tid];
	            int newDist = du + w;

	            if ((temp_distance[nid] > newDist) && (du != INT_MAX))
	            {
	            	atomicExch(&temp_distance[nid], newDist);
	            }
	        }
	    }
    }
}

__global__ void cudainitDistance(int *distance, unsigned int *temp_distance, int *visited, 
	                             int *nv, int *max_distance)
{
	int bdim = blockDim.x, gdim = gridDim.x, bid = blockIdx.x, tid = threadIdx.x;
	int i = bdim * bid + tid;
	int skip = bdim * gdim;

	for (int k = i; k < *nv; k += skip) 
	{
		distance[k] = *max_distance; //instead of INT_MAX, calculated possible max_distance
		temp_distance[k] = *max_distance;
		visited[k] = 0;
	}

	__syncthreads();

}


bool visitempty(int *visited, int count)
{
    for(int i = 0; i < count; i++ )
    {
        if (visited[i] == 1)

            return false;
    }

    return true;
}



extern "C"


void apprshybrid(const int *row_ptr, const int *col_ind, const int *weights, int **distance, 
	             int **previous, const int nv, const int ne, int source, int neg_edge_count, 
	             float **appr_vals, int max_distance, float *time)
{
	// Initialize GPU variables
	int *d_row_ptr, *d_col_ind, *d_weights, *d_distance, *d_previous, *d_visited, *d_nv, *d_ne,
	    *d_max_distance, *d_min_edges;
	unsigned int *d_temp_distance;

	
	// Initialize CPU variables
	*distance = (int*)malloc(nv*sizeof(int)); 
	*previous = (int*)malloc(nv*sizeof(int));
	int *visited = (int*)calloc(nv, sizeof(int));
	int *temp_distance = (int*)malloc(nv*sizeof(int));

	
	//SIGNALSs
	int signal_partial_graph_process = (*appr_vals)[0];
	int signal_reduce_execution = (*appr_vals)[1];
	int iter_num = (*appr_vals)[2];
	float *percentage = (float*)malloc(nv*sizeof(float));
	*percentage = (*appr_vals)[3];
	int min_edges = (*appr_vals)[4];


	// Allocate device
	cudaCheck(hipMalloc((void **)&d_row_ptr, (nv+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_col_ind, (ne+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_weights, (ne+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_distance, nv*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_previous, nv*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_visited, (nv+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_nv, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_ne, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_temp_distance, nv*sizeof(unsigned int)));
	cudaCheck(hipMalloc((void **)&d_max_distance, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_min_edges, sizeof(int)));

	//Copy inputs to device
	cudaCheck(hipMemcpy(d_row_ptr, row_ptr, (nv+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_col_ind, col_ind, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_weights, weights, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
	//cudaCheck(hipMemcpy(d_distance, *distance, nv*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_nv, &nv, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_ne, &ne, sizeof(int), hipMemcpyHostToDevice));
	//cudaCheck(hipMemcpy(d_visited, visited, (nv+1)*sizeof(int), hipMemcpyHostToDevice));
	//cudaCheck(hipMemcpy(d_temp_distance, temp_distance, nv*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_max_distance, &max_distance, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_min_edges, &min_edges, sizeof(int), hipMemcpyHostToDevice));


	hipEvent_t start;
	hipEvent_t stop;

	cudaCheck(hipEventCreate(&start));
	cudaCheck(hipEventCreate(&stop));
	cudaCheck(hipEventRecord(start, 0));

	int threadnum = 128;
	int gridSize = (nv + threadnum - 1) / threadnum; 


	cudainitDistance<<<gridSize, threadnum>>>(d_distance, d_temp_distance, d_visited, d_nv, d_max_distance);

	cudaCheck(hipMemcpy(*distance, d_distance, nv*sizeof(int), hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(temp_distance, d_temp_distance, nv*sizeof(unsigned int), hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(visited, d_visited, nv*sizeof(int), hipMemcpyDeviceToHost));


	(*distance)[source] = 0;
	visited[source] = 1;

	cudaCheck(hipMemcpy(d_distance, *distance, nv*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_visited, visited, nv*sizeof(int), hipMemcpyHostToDevice));


	int k = neg_edge_count;

	int count = 0;

	printf("neg_edge_count: %i\n", neg_edge_count);
	//printf("count: %i, k: , signal_reduce_execution: , signal_partial_graph_process: , min_edges: \n", count);

	// no approximation
	while((count) != (k+2) && !signal_reduce_execution && !signal_partial_graph_process && !min_edges)
    {
    	 while(!visitempty(visited, nv))
    	 {
	        // execute the kernel
	        cudaHSSSPKernel1<<< gridSize, threadnum >>>( d_row_ptr, d_col_ind, d_weights,
	                                                d_visited, d_distance, d_temp_distance,
	                                                nv, ne );


	        cudaHSSSPKernel2<<< gridSize, threadnum >>>( d_row_ptr, d_col_ind, d_weights,
	                                                d_visited, d_distance, d_temp_distance );

	        cudaCheck(hipMemcpy( visited, d_visited, sizeof(int) * (nv+1), hipMemcpyDeviceToHost ));
        }

        (count)++;
    
    }


    //min edges to process signal is active. only some edges are processed in kernel
    while((count) != (k+2) && !signal_reduce_execution && !signal_partial_graph_process && min_edges)
    {
    	 while(!visitempty(visited, nv))
    	 {
	        // execute the kernel
	        cudaHSSSPKernel1MoreEdges<<< gridSize, threadnum >>> ( d_row_ptr, d_col_ind, d_weights,
                                   							  d_visited, d_distance, d_temp_distance,
                                   							  nv, ne, d_min_edges );
	       

	        cudaHSSSPKernel2<<< gridSize, threadnum >>>( d_row_ptr, d_col_ind, d_weights,
	                                                d_visited, d_distance, d_temp_distance );
	        

	        cudaCheck(hipMemcpy( visited, d_visited, sizeof(int) * (nv+1), hipMemcpyDeviceToHost ));
	       
        }

        (count)++;
    
    }


    while((count) != (k+2) && signal_reduce_execution && !signal_partial_graph_process && !min_edges)
    {
    	 while(!visitempty(visited, nv))
    	 {
	        // execute the kernel
	        cudaHSSSPKernel1<<< gridSize, threadnum >>>( d_row_ptr, d_col_ind, d_weights,
	                                                d_visited, d_distance, d_temp_distance,
	                                                nv, ne );
	       

	        cudaHSSSPKernel2<<< gridSize, threadnum >>>( d_row_ptr, d_col_ind, d_weights,
	                                                d_visited, d_distance, d_temp_distance );
	        

	        cudaCheck(hipMemcpy( visited, d_visited, sizeof(int) * (nv+1), hipMemcpyDeviceToHost ));
	       
        }

        (count)++;
    
    }

	//Copy outputs to host
	cudaCheck(hipMemcpy(*distance, d_distance, nv*sizeof(int), hipMemcpyDeviceToHost));
	
	cudaCheck(hipEventRecord(stop, 0));
	cudaCheck(hipEventSynchronize(stop));
	float elapsed;
	cudaCheck(hipEventElapsedTime(&elapsed, start, stop));


	printf("count: %i\n", count);


	// Deallocation
	cudaCheck(hipFree(d_row_ptr));
	cudaCheck(hipFree(d_col_ind));
	cudaCheck(hipFree(d_weights));
	cudaCheck(hipFree(d_distance));
	cudaCheck(hipFree(d_previous));

	printf("GPU SHYBRID time: %f\n", elapsed);

	*time = elapsed;

	
}