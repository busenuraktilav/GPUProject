#include "hip/hip_runtime.h"

extern "C" {
#include "bellman_ford.cuh"
}

#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <iostream>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <omp.h>

#define N_THREADS_PER_BLOCK 128


#define cudaCheck(error) \
	if (error != hipSuccess) { \
    	printf("Fatal error: %s at %s:%d\n", \
    		hipGetErrorString(error), \
    		__FILE__, __LINE__); \
    	exit(1); \
 }

__global__ void cudaRelax(int *row_ptr, int *col_ind, int *weights, int *queue, int *nextQueue, int size, int* nextSize, int* nv, int* distance) 
{

    int index, u, v, w, du, dv, add, tid = threadIdx.x + (blockDim.x * blockIdx.x);

    if (tid < size) { 
        
    	u = queue[tid];
    	du = distance[u];

        for (int e = row_ptr[u]; e < row_ptr[u + 1]; e++) {
            
            v = col_ind[e];
            w = weights[e];
            dv = distance[v];
            add = du + w;

            if (add < dv && du != INT_MAX)
            {
            	atomicMin(&distance[v],add);
            	index = atomicAdd(nextSize, 1);
				nextQueue[index] = v;
            }

        }
    }

}

__global__ void cudarelaxAtom(int *row_ptr, int *col_ind, int *weights, int *queue, int *nextQueue, int size, int* nextSize, int2* distance, int *iter) 
{

    int index, u, v, w, du, dv, add, tid = threadIdx.x + (blockDim.x * blockIdx.x);

    if (tid < size) { 
        
    	u = queue[tid];
    	du = distance[u].x;

        for (int e = row_ptr[u]; e < row_ptr[u + 1]; e++) {
            
            v = col_ind[e];
            w = weights[e];
            dv = distance[v].x;
            add = du + w;

            if (add < dv && du != INT_MAX)
            {
            	atomicMin(&distance[v].x,add);

            	if(distance[v].y != *iter)
            	{
	            	atomicMax(&distance[v].y,*iter);
	            	index = atomicAdd(nextSize, 1);
					nextQueue[index] = v;
				}
            }
        }
    }
}



__global__ void cudarelaxAtomicAddBlock(int *row_ptr, int *col_ind, int *weights, int *queue, int *nextQueue, int size, int* nextSize, int2* distance, int *iter) 
{

    int index, u, v, w, du, dv, add, tid = threadIdx.x + (blockDim.x * blockIdx.x);

    if (tid < size) { 
        
    	u = queue[tid];
    	du = distance[u].x;

        for (int e = row_ptr[u]; e < row_ptr[u + 1]; e++) {
            
            v = col_ind[e];
            w = weights[e];
            dv = distance[v].x;
            add = du + w;

            if (add < dv && du != INT_MAX)
            {
            	atomicMin(&distance[v].x,add);

            	if(distance[v].y != *iter)
            	{
	            	atomicMax(&distance[v].y,*iter);
	            	index = atomicAdd_block(nextSize, 1);
					nextQueue[index] = v;
				}
            }
        }
    }
}


__global__ void cudarelaxAtomicMinBlock(int *row_ptr, int *col_ind, int *weights, int *queue, int *nextQueue, int size, int* nextSize, int2* distance, int *iter) 
{

    int index, u, v, w, du, dv, add, tid = threadIdx.x + (blockDim.x * blockIdx.x);

    if (tid < size) { 
        
    	u = queue[tid];
    	du = distance[u].x;

        for (int e = row_ptr[u]; e < row_ptr[u + 1]; e++) {
            
            v = col_ind[e];
            w = weights[e];
            dv = distance[v].x;
            add = du + w;

            if (add < dv && du != INT_MAX)
            {
            	atomicMin_block(&distance[v].x,add);

            	if(distance[v].y != *iter)
            	{
	            	atomicMax(&distance[v].y,*iter);
	            	index = atomicAdd(nextSize, 1);
					nextQueue[index] = v;
				}
            }
        }
    }
}


__global__ void cudarelaxAtomicMaxBlock(int *row_ptr, int *col_ind, int *weights, int *queue, int *nextQueue, int size, int* nextSize, int2* distance, int *iter) 
{

    int index, u, v, w, du, dv, add, tid = threadIdx.x + (blockDim.x * blockIdx.x);

    if (tid < size) { 
        
    	u = queue[tid];
    	du = distance[u].x;

        for (int e = row_ptr[u]; e < row_ptr[u + 1]; e++) {
            
            v = col_ind[e];
            w = weights[e];
            dv = distance[v].x;
            add = du + w;

            if (add < dv && du != INT_MAX)
            {
            	atomicMin(&distance[v].x,add);

            	if(distance[v].y != *iter)
            	{
	            	atomicMax_block(&distance[v].y,*iter);
	            	index = atomicAdd(nextSize, 1);
					nextQueue[index] = v;
				}
            }
        }
    }
}



__global__ void cudarelaxAtomicMoreEdges(int *row_ptr, int *col_ind, int *weights, int *queue, 
	                                     int *nextQueue, int size, int* nextSize, int2* distance, 
	                                     int *iter, int *min_edges) 
{

    int index, u, v, w, du, dv, add, tid = threadIdx.x + (blockDim.x * blockIdx.x);

    if (tid < size) { 

    	u = queue[tid];
    	du = distance[u].x;

    	if ((row_ptr[u+1] - row_ptr[u]) > *min_edges) // process only the nodes which has more than ... edges 
    	{
	        for (int e = row_ptr[u]; e < row_ptr[u + 1]; e++) {
	            
	            v = col_ind[e];
	            w = weights[e];
	            dv = distance[v].x;
	            add = du + w;

	            if (add < dv && du != INT_MAX)
	            {
	            	atomicMin(&distance[v].x,add);

	            	if(distance[v].y != *iter)
	            	{
		            	atomicMax(&distance[v].y,*iter);
		            	index = atomicAdd(nextSize, 1);
		            	//printf("nextSize: %i\n", nextSize);
						nextQueue[index] = v;
					}
	            }
	        }
	    }
    }
}

__global__ void cudainitVar(int2 *distance, int *nv, int *max_distance)
{
	int bdim = blockDim.x, gdim = gridDim.x, bid = blockIdx.x, tid = threadIdx.x;
	int i = bdim * bid + tid;
	int skip = bdim * gdim;

	for (int k = i; k < *nv; k += skip) {

		distance[k].x = *max_distance; //instead of INT_MAX, calculated possible max_distance
		distance[k].y = 1; // dont update this in relaxinit. update here. no need for mem copy
	}

	__syncthreads();

}


__global__ void cudasubset_of_vertices(int *size, float *percentage, int *queue, int *subset_queue, unsigned int *random_ints)
{
	int i = threadIdx.x + (blockDim.x * blockIdx.x);

	int t = (*size) * (*percentage);
	int temp = t;
	if (i == 1)
	{
		printf("temp: %i, size: %i, percentage: %f\n", temp, *size, *percentage);
	}
	                     

	if (i < t)
	{
		subset_queue[i] = (queue)[random_ints[i]%(t)];
		if ((*percentage) == 0.1 )
		{
			printf("subset_queue[%i]: %i , (queue)[random_ints[%i]=%i % (t=%i)]: %i\n", i, subset_queue[i], i, random_ints[i], t, (queue)[random_ints[i]%(t)]);
		}
		
	}

	(*size) = temp;

}


extern "C"

void apprbf(const int *row_ptr, const int *col_ind, const int *weights, int **distance, 
	        const int nv, const int ne, int source, int *signals, float **signal_variables, 
	        int max_distance, float *time)
{

	// Initialize GPU variables
	int *d_row_ptr, *d_col_ind, *d_weights, *d_nv, *d_max_distance;
	int *d_queue, *d_nextQueue, *d_nextSize, *d_iter, *d_subset_queue, *d_min_edges;
	float *d_percentage;
	int2* d_dist;
	

	int2* dist = (int2*)malloc(nv*sizeof(int2));

	
	//CUDA RANDOM GENERATOR
	hiprandGenerator_t gen;
	unsigned int *d_random_ints;
	cudaCheck(hipMalloc((void **)&d_random_ints, nv*sizeof(unsigned int)));
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT); //Create pseudo-random number generator
	    

	//SIGNALS
	int signal_partial_graph_process = signals[4];
	int signal_reduce_execution = signals[3];
	int iter_num = (*signal_variables)[1];
	float *percentage = (float*) malloc (sizeof(float));
	*percentage = (*signal_variables)[2];
	int min_edges = (*signal_variables)[0];
	int signal_atomicMinBlock = signals[5];
	int signal_atomicMaxBlock = signals[6];
	int signal_atomicAddBlock = signals[7];


	cudaCheck(hipMalloc((void **)&d_row_ptr, (nv+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_col_ind, (ne+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_weights, (ne+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_dist, nv*sizeof(int2)));
	cudaCheck(hipMalloc((void **)&d_nv, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_max_distance, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_subset_queue, (nv+1)*sizeof(int))); //space might be percentage*size 
	cudaCheck(hipMalloc((void **)&d_min_edges, sizeof(int))); //for kernel cudarelaxAtomicMoreEdges, min edges

	cudaCheck(hipMemcpy(d_nv, &nv, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_max_distance, &max_distance, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_min_edges, &min_edges, sizeof(int), hipMemcpyHostToDevice));


	cudainitVar<<<(nv + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_dist, d_nv, d_max_distance);

	cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));


	float elapsed = 0.0, elapsed0 = 0.0;

	hipEvent_t start;
	hipEvent_t stop;
	cudaCheck(hipEventCreate(&start));
	cudaCheck(hipEventCreate(&stop));
	cudaCheck(hipEventRecord(start, 0));

	//To increase parallelism, first process the source vertex
	int srcNeigh = row_ptr[source + 1] - row_ptr[source];
	int *srcArr = (int*)calloc(srcNeigh, sizeof(int));
	int index = 0;
	dist[source].x = 0;

	for (int i = row_ptr[source]; i < row_ptr[source + 1]; i++) { // for each neighbor of source node
		
		int w = weights[i];
		int du = dist[source].x;
		int dv = dist[col_ind[i]].x;

		if (du + w < dv && du != INT_MAX)
		{
			dist[col_ind[i]].x = du + w;
			dist[col_ind[i]].y = 1;
			srcArr[index++] = col_ind[i]; // add to frontier
		}
	}

	cudaCheck(hipEventRecord(stop, 0));
	cudaCheck(hipEventSynchronize(stop));
	cudaCheck(hipEventElapsedTime(&elapsed0, start, stop));


	int *iter = (int*)malloc(sizeof(int));
	*iter = 2;

	// Allocate device
	
	cudaCheck(hipMalloc((void **)&d_nextSize, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_queue, (nv+1)*sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_nextQueue, (nv+1)*sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_iter, sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_percentage, sizeof(float)));
    
	//Copy inputs to device
	cudaCheck(hipMemcpy(d_row_ptr, row_ptr, (nv+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_col_ind, col_ind, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_weights, weights, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_dist, dist, nv*sizeof(int2), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_queue, srcArr, srcNeigh * sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_percentage, percentage, sizeof(float), hipMemcpyHostToDevice));

	int size = srcNeigh;
	int *nextSize = (int*)malloc(sizeof(int));
	*nextSize = 0;


	int round = 1;
	int temp = 0;

	cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));

	if (!signal_reduce_execution && !signal_partial_graph_process && !min_edges && !signal_atomicMinBlock && !signal_atomicMaxBlock && !signal_atomicAddBlock)
	{
		hipEvent_t start;
		hipEvent_t stop;
		cudaCheck(hipEventCreate(&start));
		cudaCheck(hipEventCreate(&stop));
		cudaCheck(hipEventRecord(start, 0));

		//hipProfilerStart();

		while((size > 0) && (round < nv) && temp < ne) { temp += size;

			//printf("NO APPR\n");

			cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
			cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));

			cudarelaxAtom<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_row_ptr, d_col_ind, d_weights, d_queue, d_nextQueue, size, d_nextSize, d_dist, d_iter); 

			cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));

			(*iter) ++;
			printf("size: %i\n", size);
			size = *nextSize;
			*nextSize = 0;
			std::swap(d_queue, d_nextQueue); // swap frontiers

			(*signal_variables)[1] = round;
			round++;

		}
		
		//hipProfilerStop();

		cudaCheck(hipEventRecord(stop, 0));
		cudaCheck(hipEventSynchronize(stop));
		cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

		// Copy outputs to host
		cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));
	}


	if (min_edges && !signal_partial_graph_process && !signal_reduce_execution)
	{
		//min edges to process signal is active. only some edges are processed in kernel

		hipEvent_t start;
		hipEvent_t stop;
		cudaCheck(hipEventCreate(&start));
		cudaCheck(hipEventCreate(&stop));
		cudaCheck(hipEventRecord(start, 0));

		//hipProfilerStart();

		while((size > 0) && (round < nv) && temp < ne) { temp += size;

			cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
			cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));

			cudarelaxAtomicMoreEdges<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_row_ptr, d_col_ind, d_weights, d_queue, d_nextQueue, size, d_nextSize, d_dist, d_iter, d_min_edges); 

			cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));

			(*iter) ++;
			printf("size: %i\n", size);
			size = *nextSize;
			*nextSize = 0;
			std::swap(d_queue, d_nextQueue); // swap frontiers
		}

		//hipProfilerStop();

		cudaCheck(hipEventRecord(stop, 0));
		cudaCheck(hipEventSynchronize(stop));
		cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

		// Copy outputs to host
		cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));
	}

	
	if (signal_partial_graph_process && !min_edges && !signal_reduce_execution)
	{
		// If reduce signal is negative && partial graph processing signal positive

		hipEvent_t start;
		hipEvent_t stop;
		cudaCheck(hipEventCreate(&start));
		cudaCheck(hipEventCreate(&stop));
		cudaCheck(hipEventRecord(start, 0));

		//hipProfilerStart();

		while((size > 0) && (round < nv) && temp < ne) { temp += size;

			cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
			cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));

			cudarelaxAtom<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_row_ptr, d_col_ind, d_weights, d_queue, d_nextQueue, size, d_nextSize, d_dist, d_iter);

			cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));


			(*iter) ++;	
			std::swap(d_queue, d_nextQueue); // swap frontiers
			hiprandGenerate(gen, d_random_ints, size); //Generate n ints on device

			cudasubset_of_vertices<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_nextSize, d_percentage, d_queue, d_subset_queue, d_random_ints);

			cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));
			
			std::swap(d_queue, d_subset_queue);
			printf("size: %i\n", size);		
			size = *nextSize;
			*nextSize = 0;

			round++;

		}

		//hipProfilerStop();

		cudaCheck(hipEventRecord(stop, 0));
		cudaCheck(hipEventSynchronize(stop));
		
		cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

		// Copy outputs to host
		cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));
	}
	
	
	if (signal_reduce_execution && !signal_partial_graph_process && !min_edges)
	{
		//If reduce signal is positive && partial process signal negative

		hipEvent_t start;
		hipEvent_t stop;
		cudaCheck(hipEventCreate(&start));
		cudaCheck(hipEventCreate(&stop));
		cudaCheck(hipEventRecord(start, 0));

		//hipProfilerStart();

		while((round < iter_num+1) ) { temp += size;

			cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
			cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));

			cudarelaxAtom<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_row_ptr, d_col_ind, d_weights, d_queue, d_nextQueue, size, d_nextSize, d_dist, d_iter);

			cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));

			(*iter) ++;
			printf("size: %i\n", size);
			size = *nextSize;
			*nextSize = 0;
			std::swap(d_queue, d_nextQueue); // swap frontiers
			round++;

		}

		//hipProfilerStop();

		cudaCheck(hipEventRecord(stop, 0));
		cudaCheck(hipEventSynchronize(stop));
		
		cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

		// Copy outputs to host
		cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));
	}


	if (signal_reduce_execution && min_edges && !signal_partial_graph_process)
	{
		//If reduce signal is positive && partial process signal negative

		hipEvent_t start;
		hipEvent_t stop;
		cudaCheck(hipEventCreate(&start));
		cudaCheck(hipEventCreate(&stop));
		cudaCheck(hipEventRecord(start, 0));

		//hipProfilerStart();

		while((round < iter_num+1) ) { temp += size;

			cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
			cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));

			cudarelaxAtomicMoreEdges<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_row_ptr, d_col_ind, d_weights, d_queue, d_nextQueue, size, d_nextSize, d_dist, d_iter, d_min_edges); 

			cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));

			(*iter) ++;
			printf("size: %i\n", size);
			size = *nextSize;
			*nextSize = 0;
			std::swap(d_queue, d_nextQueue); // swap frontiers
			round++;

		}
		
		//hipProfilerStop();

		cudaCheck(hipEventRecord(stop, 0));
		cudaCheck(hipEventSynchronize(stop));
		
		cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

		// Copy outputs to host
		cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));
	}

	if (signal_reduce_execution && signal_partial_graph_process && !min_edges)
	{
		//If reduce signal is positive && partial process signal negative

		hipEvent_t start;
		hipEvent_t stop;
		cudaCheck(hipEventCreate(&start));
		cudaCheck(hipEventCreate(&stop));
		cudaCheck(hipEventRecord(start, 0));

		//hipProfilerStart();

		while((round < iter_num+1) ) { temp += size;

			cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
			cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));

			cudarelaxAtom<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_row_ptr, d_col_ind, d_weights, d_queue, d_nextQueue, size, d_nextSize, d_dist, d_iter);

			cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));

			(*iter) ++;
			std::swap(d_queue, d_nextQueue); // swap frontiers
			hiprandGenerate(gen, d_random_ints, size); //Generate n ints on device

			cudasubset_of_vertices<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_nextSize, d_percentage, d_queue, d_subset_queue, d_random_ints);

			cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));

			std::swap(d_queue, d_subset_queue);
			printf("size: %i\n", size);	
			size = *nextSize;
			*nextSize = 0;

			round++;
		}

		//hipProfilerStop();

		cudaCheck(hipEventRecord(stop, 0));
		cudaCheck(hipEventSynchronize(stop));
		
		cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

		// Copy outputs to host
		cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));
	}


	if (signal_partial_graph_process && min_edges && !signal_reduce_execution)
	{
		// If reduce signal is negative && partial graph processing signal positive

		hipEvent_t start;
		hipEvent_t stop;
		cudaCheck(hipEventCreate(&start));
		cudaCheck(hipEventCreate(&stop));
		cudaCheck(hipEventRecord(start, 0));

		//hipProfilerStart();

		while((size > 0) && (round < nv) && temp < ne) { temp += size;

			cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
			cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));

			cudarelaxAtomicMoreEdges<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_row_ptr, d_col_ind, d_weights, d_queue, d_nextQueue, size, d_nextSize, d_dist, d_iter, d_min_edges); 
			
			cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));

			(*iter) ++;
			std::swap(d_queue, d_nextQueue); // swap frontiers
			hiprandGenerate(gen, d_random_ints, size); //Generate n ints on device

			cudasubset_of_vertices<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_nextSize, d_percentage, d_queue, d_subset_queue, d_random_ints);

			cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));

			std::swap(d_queue, d_subset_queue);
			printf("size: %i\n", size);	
			size = *nextSize;
			*nextSize = 0;

			(*signal_variables)[1] = round;
			round++;

		}

		//hipProfilerStop();

		cudaCheck(hipEventRecord(stop, 0));
		cudaCheck(hipEventSynchronize(stop));
		
		cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

		// Copy outputs to host
		cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));
	}


	if (signal_partial_graph_process && min_edges && signal_reduce_execution)
	{
		// If reduce signal is negative && partial graph processing signal positive

		hipEvent_t start;
		hipEvent_t stop;
		cudaCheck(hipEventCreate(&start));
		cudaCheck(hipEventCreate(&stop));
		cudaCheck(hipEventRecord(start, 0));

		//hipProfilerStart();

		while((round < iter_num+1) ) { temp += size;

			cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
			cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));

			cudarelaxAtomicMoreEdges<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_row_ptr, d_col_ind, d_weights, d_queue, d_nextQueue, size, d_nextSize, d_dist, d_iter, d_min_edges); 
			
			cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));

			(*iter) ++;
			std::swap(d_queue, d_nextQueue); // swap frontiers
			hiprandGenerate(gen, d_random_ints, size); //Generate n ints on device

			cudasubset_of_vertices<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_nextSize, d_percentage, d_queue, d_subset_queue, d_random_ints);

			cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));

			std::swap(d_queue, d_subset_queue);
			printf("size: %i\n", size);		
			size = *nextSize;
			*nextSize = 0;
			round++;

		}

		//hipProfilerStop();

		cudaCheck(hipEventRecord(stop, 0));
		cudaCheck(hipEventSynchronize(stop));
		
		cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

		// Copy outputs to host
		cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));
	}

	
	
	if (signal_atomicAddBlock)
	{
		hipEvent_t start;
		hipEvent_t stop;
		cudaCheck(hipEventCreate(&start));
		cudaCheck(hipEventCreate(&stop));
		cudaCheck(hipEventRecord(start, 0));

		//hipProfilerStart();
	
		while((size > 0) && (round < nv) && temp < ne) { temp += size;

			cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
			cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));

			cudarelaxAtomicAddBlock<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_row_ptr, d_col_ind, d_weights, d_queue, d_nextQueue, size, d_nextSize, d_dist, d_iter); 

			cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));

			(*iter) ++;
			printf("size: %i\n", size);
			size = *nextSize;
			*nextSize = 0;
			std::swap(d_queue, d_nextQueue); // swap frontiers
			round++;

		}

		//hipProfilerStop();

		cudaCheck(hipEventRecord(stop, 0));
		cudaCheck(hipEventSynchronize(stop));
		cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

		// Copy outputs to host
		cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));
	}

	
	if (signal_atomicMinBlock)
	{
		hipEvent_t start;
		hipEvent_t stop;
		cudaCheck(hipEventCreate(&start));
		cudaCheck(hipEventCreate(&stop));
		cudaCheck(hipEventRecord(start, 0));

		//hipProfilerStart();

		while((size > 0) && (round < nv) && temp < ne) { temp += size;

			cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
			cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));

			cudarelaxAtomicMinBlock<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_row_ptr, d_col_ind, d_weights, d_queue, d_nextQueue, size, d_nextSize, d_dist, d_iter); 

			cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));

			(*iter) ++;
			printf("size: %i\n", size);
			size = *nextSize;
			*nextSize = 0;
			std::swap(d_queue, d_nextQueue); // swap frontiers
			round++;
		}

		//hipProfilerStop();

		cudaCheck(hipEventRecord(stop, 0));
		cudaCheck(hipEventSynchronize(stop));
		cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

		// Copy outputs to host
		cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));
	}


	if (signal_atomicMaxBlock)
	{
		hipEvent_t start;
		hipEvent_t stop;
		cudaCheck(hipEventCreate(&start));
		cudaCheck(hipEventCreate(&stop));
		cudaCheck(hipEventRecord(start, 0));

		//hipProfilerStart();

		while((size > 0) && (round < nv) && temp < ne) { temp += size;

			cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
			cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));

			cudarelaxAtomicMaxBlock<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_row_ptr, d_col_ind, d_weights, d_queue, d_nextQueue, size, d_nextSize, d_dist, d_iter); 

			cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));

			(*iter) ++;
			printf("size: %i\n", size);
			size = *nextSize;
			*nextSize = 0;
			std::swap(d_queue, d_nextQueue); // swap frontiers
			round++;
		}

		//hipProfilerStop();

		cudaCheck(hipEventRecord(stop, 0));
		cudaCheck(hipEventSynchronize(stop));
		cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

		// Copy outputs to host
		cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));
	}

	

	printf("GPU SBF time(ms): %f\n", elapsed+elapsed0);
	*time = elapsed+elapsed0;

	printf("elapsed: %f, elapsed0: %f, add: %f\n", elapsed, elapsed0, elapsed0+elapsed);
	

	// check for negative cycles
	int neg_cycle = false;

	for (int v2 = 0; v2 < nv; v2++)
	{
		for (int e = (row_ptr)[v2]; e < (row_ptr)[v2+1]; e++)
		{
			int tempDistance = dist[v2].x + (weights)[e];
			int neigh = (col_ind)[e];

			if ((tempDistance < dist[neigh].x) && (dist[v2].x != INT_MAX))
			{
				printf("GPU belman ford Error: negative cycle exists\n");
				neg_cycle = true;
				break;
			}

		}
		if(neg_cycle)
			break;
	}

	printf("NEGATIVE CYCLE CHECK: %i\n", neg_cycle);


	*distance = (int*)malloc((nv+1)*sizeof(int)); 

	for (int i = 0; i < nv; i++)
	{
		(*distance)[i] = dist[i].x;
	}


	// Deallocation
	cudaCheck(hipFree(d_row_ptr));
	cudaCheck(hipFree(d_col_ind));
	cudaCheck(hipFree(d_weights));
	cudaCheck(hipFree(d_dist));
	cudaCheck(hipFree(d_nv));
	cudaCheck(hipFree(d_max_distance));
	cudaCheck(hipFree(d_queue));
	cudaCheck(hipFree(d_nextQueue));
	cudaCheck(hipFree(d_nextSize));
	cudaCheck(hipFree(d_iter));
	cudaCheck(hipFree(d_subset_queue));
	cudaCheck(hipFree(d_min_edges));
}