#include "hip/hip_runtime.h"

extern "C" {
#include "bellman_ford.cuh"
}

#include <stdio.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define INF 9999
#define N_THREADS_PER_BLOCK 1024


#define cudaCheck(error) \
	if (error != hipSuccess) { \
    	printf("Fatal error: %s at %s:%d\n", \
    		hipGetErrorString(error), \
    		__FILE__, __LINE__); \
    	exit(1); \
 }

__global__ void cudasbf(int *row_ptr, int *col_ind, int *row_ind, int *weights, int *distance, int *previous, int nv, int ne)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;

	for(int v = 0; v < nv; v++)
	{
		if(tid < ne)
		{
			//printf("%i\n", tid);
			int tempdistance = distance[(row_ind)[tid]] + weights[tid];

			if(tempdistance < distance[(col_ind)[tid]] && distance[(row_ind)[tid]] != INT_MAX)
			{
				distance[(col_ind)[tid]] = tempdistance;
				previous[(col_ind)[tid]] = row_ind[tid];
			}
		}
	}
}


extern "C"

void sbf(const int *row_ptr, const int *col_ind, const int *row_ind, const int *weights, int **distance, int **previous, const int nv, const int ne, int source)
{
	// Initialize GPU variables
	int *d_row_ptr, *d_col_ind, *d_row_ind, *d_weights, *d_distance, *d_previous, *d_nv, *d_ne;
	
	// Initialize CPU variables
	*distance = (int*)malloc(nv*sizeof(int)); 
	*previous = (int*)malloc(nv*sizeof(int));

	for (int i = 0; i < nv; i++)
	{
		(*distance)[i] = INT_MAX;
		(*previous)[i] = -1;
	}

	(*distance)[source] = 0;


	// Allocate device
	hipMalloc((void **)&d_row_ptr, (nv+1)*sizeof(int));
	hipMalloc((void **)&d_col_ind, (ne+1)*sizeof(int));
	hipMalloc((void **)&d_row_ind, (ne+1)*sizeof(int));
	hipMalloc((void **)&d_weights, (ne+1)*sizeof(int));
	hipMalloc((void **)&d_distance, nv*sizeof(int));
	hipMalloc((void **)&d_previous, nv*sizeof(int));
	hipMalloc((void **)&d_nv, sizeof(int));
	hipMalloc((void **)&d_ne, sizeof(int));

	//Copy inputs to device
	hipMemcpy(d_row_ptr, row_ptr, (nv+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_col_ind, col_ind, (ne+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_row_ind, row_ind, (ne+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_weights, weights, (ne+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_distance, (*distance), nv*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_nv, &nv, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_ne, &ne, sizeof(int), hipMemcpyHostToDevice);


	hipEvent_t start;
	hipEvent_t stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	cudasbf<<<(ne+1023)/1024, 1024>>>(d_row_ptr, d_col_ind, d_row_ind, d_weights, d_distance, d_previous, nv, ne);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsed;
	hipEventElapsedTime(&elapsed, start, stop);

	// Copy outputs to host
	hipMemcpy((*distance), d_distance, nv*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy((*previous), d_previous, nv*sizeof(int), hipMemcpyDeviceToHost);


	// check for negative cycles
	for(int e = 0; e < ne; e++)
	{
		if((*distance)[row_ind[e]] + weights[e] < (*distance)[(col_ind)[e]] && (*distance)[row_ind[e]] != INT_MAX)
		{
			printf("cuda Error: negative cycle exists\n");
			break;
		}
	}



	// Deallocation
	hipFree(d_row_ptr);
	hipFree(d_col_ind);
	hipFree(d_row_ind);
	hipFree(d_weights);
	hipFree(d_distance);
	hipFree(d_previous);

	printf("GPU SBF time: %f\n", elapsed/1000);
}