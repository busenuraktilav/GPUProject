#include "hip/hip_runtime.h"

extern "C" {
#include "bellman_ford.cuh"
}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define INF 9999
#define N_THREADS_PER_BLOCK 1024


#define cudaCheck(error) \
	if (error != hipSuccess) { \
    	printf("Fatal error: %s at %s:%d\n", \
    		hipGetErrorString(error), \
    		__FILE__, __LINE__); \
    	exit(1); \
 }

__global__ void cudasbf(int *row_ptr, int *col_ind, int *row_ind, int *weights, int *distance, int *previous, int nv, int ne)
{
	const int tid = blockIdx.x * blockDim.x + threadIdx.x;

	for(int v = 0; v < nv; v++)
	{
		if(tid < ne)
		{
			int tempdistance = distance[(row_ind)[tid]] + weights[tid];

			if(tempdistance < distance[(col_ind)[tid]])
			{
				distance[(col_ind)[tid]] = tempdistance;
				previous[(col_ind)[tid]] = row_ind[tid];
			}
		}
	}
}


extern "C"

void sbf(const int *row_ptr, const int *col_ind, const int *row_ind, const int *weights, int *distance, int *previous, const int nv, const int ne, int source)
{
	// Initialize GPU variables
	int *d_row_ptr;
	int *d_col_ind;
	int *d_row_ind;
	int *d_weights;
	int *d_distance;
	int *d_previous;
	int *d_nv;
	int *d_ne;
	
	// Initialize CPU variables
	distance[source] = 0;


	//int snv = (nv+1)*sizeof(int);
	//int sne = (ne+1)*sizeof(int);

	// Allocate device
	hipMalloc((void **)&d_row_ptr, (nv+1)*sizeof(int));
	hipMalloc((void **)&d_col_ind, (ne+1)*sizeof(int));
	hipMalloc((void **)&d_row_ind, (ne+1)*sizeof(int));
	hipMalloc((void **)&d_weights, (ne+1)*sizeof(int));
	hipMalloc((void **)&d_distance, nv*sizeof(int));
	hipMalloc((void **)&d_previous, nv*sizeof(int));
	hipMalloc((void **)&d_nv, sizeof(int));
	hipMalloc((void **)&d_ne, sizeof(int));

	//Copy inputs to device
	hipMemcpy(d_row_ptr, row_ptr, (nv+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_col_ind, col_ind, (ne+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_row_ind, row_ind, (ne+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_weights, weights, (ne+1)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_distance, distance, nv*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_nv, &nv, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_ne, &ne, sizeof(int), hipMemcpyHostToDevice);


	hipEvent_t start;
	hipEvent_t stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	cudasbf<<<(ne+1023)/1024, 1024>>>(d_row_ptr, d_col_ind, d_row_ind, d_weights, d_distance, d_previous, nv, ne);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsed;
	hipEventElapsedTime(&elapsed, start, stop);

	// Copy outputs to host
	hipMemcpy(distance, d_distance, nv*sizeof(int), hipMemcpyDeviceToHost);


	// check for negative cycles
	for(int e = 0; e < ne; e++)
	{
		if(distance[row_ind[e]] + weights[e] < distance[(col_ind)[e]])
			printf("Error: negative cycle exists\n");
	}


	// Deallocation
	hipFree(d_row_ptr);
	hipFree(d_col_ind);
	hipFree(d_row_ind);
	hipFree(d_weights);
	hipFree(d_distance);
	hipFree(d_previous);

	printf("GPU SBF time: %f\n", elapsed);
}