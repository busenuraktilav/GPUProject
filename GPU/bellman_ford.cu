#include "hip/hip_runtime.h"

extern "C" {
#include "bellman_ford.cuh"
}

#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <iostream>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define N_THREADS_PER_BLOCK 128


#define cudaCheck(error) \
	if (error != hipSuccess) { \
    	printf("Fatal error: %s at %s:%d\n", \
    		hipGetErrorString(error), \
    		__FILE__, __LINE__); \
    	exit(1); \
 }

__global__ void relax(int *row_ptr, int *col_ind, int *weights, int *queue, int *nextQueue, int size, int* nextSize, int* nv, int* distance) 
{

    int index, u, v, w, du, dv, add, tid = threadIdx.x + (blockDim.x * blockIdx.x);

    if (tid < size) { 
        
    	u = queue[tid];
    	du = distance[u];

        for (int e = row_ptr[u]; e < row_ptr[u + 1]; e++) {
            
            v = col_ind[e];
            w = weights[e];
            dv = distance[v];
            add = du + w;

            if (add < dv && du != INT_MAX)
            {
            	atomicMin(&distance[v],add);
            	index = atomicAdd(nextSize, 1);
				nextQueue[index] = v;
            }

        }
    }

}

__global__ void relaxAtom(int *row_ptr, int *col_ind, int *weights, int *queue, int *nextQueue, int size, int* nextSize, int2* distance, int *iter) 
{

    int index, u, v, w, du, dv, add, tid = threadIdx.x + (blockDim.x * blockIdx.x);

    if (tid < size) { 
        
    	u = queue[tid];
    	du = distance[u].x;

        for (int e = row_ptr[u]; e < row_ptr[u + 1]; e++) {
            
            v = col_ind[e];
            w = weights[e];
            dv = distance[v].x;
            add = du + w;

            if (add < dv && du != INT_MAX)
            {
            	atomicMin(&distance[v].x,add);

            	if(distance[v].y != *iter)
            	{
	            	atomicMax(&distance[v].y,*iter);
	            	index = atomicAdd(nextSize, 1);
					nextQueue[index] = v;
				}
            }
        }
    }
}


__global__ void initVar(int2 *distance, int *nv)
{
	int bdim = blockDim.x, gdim = gridDim.x, bid = blockIdx.x, tid = threadIdx.x;
	int i = bdim * bid + tid;
	int skip = bdim * gdim;

	for (int k = i; k < *nv; k += skip) {

		distance[k].x = INT_MAX;
		distance[k].y = 1; // dont update this in relaxinit. update here. no need for mem copy
	}

	__syncthreads();

}


__global__ void subset_of_vertices(int *size, float *percentage, int *queue, int *subset_queue)
{
	int i = threadIdx.x + (blockDim.x * blockIdx.x);

	int t = (*size) * (*percentage);
	int temp = t;
	int rand = i*t + i + t + (*percentage) * (*size);

	if (i < t)
	{
		subset_queue[i] = (queue)[rand%(t)];
		//printf("%i\n", subset_queue[i]);
	}

	(*size) = temp;
}



extern "C"

void sbf(const int *row_ptr, const int *col_ind, const int *row_ind, const int *weights, int **distance, int **previous, const int nv, const int ne, int source, float **appr_vals)
{

	// Initialize GPU variables
	int *d_row_ptr, *d_col_ind, *d_weights, *d_nv;
	int *d_queue, *d_nextQueue, *d_nextSize, *d_iter, *d_subset_queue;
	float *d_percentage;
	int2* d_dist;


	int2* dist = (int2*)malloc(nv*sizeof(int2));


	int signal_partial_graph_process = (*appr_vals)[0];
	int signal_reduce_execution = (*appr_vals)[1];
	int iter_num = (*appr_vals)[2];
	float *percentage = (float*)malloc(nv*sizeof(float));
	*percentage = (*appr_vals)[3];



	cudaCheck(hipMalloc((void **)&d_row_ptr, (nv+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_col_ind, (ne+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_weights, (ne+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_dist, nv*sizeof(int2)));
	cudaCheck(hipMalloc((void **)&d_nv, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_subset_queue, (nv+1)*sizeof(int))); //space might be percentage*size 

	cudaCheck(hipMemcpy(d_nv, &nv, sizeof(int), hipMemcpyHostToDevice));



	hipEvent_t start;
	hipEvent_t stop;
	cudaCheck(hipEventCreate(&start));
	cudaCheck(hipEventCreate(&stop));
	cudaCheck(hipEventRecord(start, 0));

	hipProfilerStart();


	initVar<<<(nv + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_dist, d_nv);

	cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));

	//To increase parallelism, first process the source vertex
	int srcNeigh = row_ptr[source + 1] - row_ptr[source];
	int *srcArr = (int*)calloc(srcNeigh, sizeof(int));
	int index = 0;
	dist[source].x = 0;

	for (int i = row_ptr[source]; i < row_ptr[source + 1]; i++) { // for each neighbor of source node
		
		int w = weights[i];
		int du = dist[source].x;
		int dv = dist[col_ind[i]].x;

		if (du + w < dv && du != INT_MAX)
		{
			dist[col_ind[i]].x = du + w;
			dist[col_ind[i]].y = 1;
			srcArr[index++] = col_ind[i]; // add to frontier
		}
	}

	int *iter = (int*)malloc(sizeof(int));
	*iter = 2;


	// Allocate device
	
	cudaCheck(hipMalloc((void **)&d_nextSize, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_queue, (nv+1)*sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_nextQueue, (nv+1)*sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_iter, sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_percentage, sizeof(float)));
    
	//Copy inputs to device
	cudaCheck(hipMemcpy(d_row_ptr, row_ptr, (nv+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_col_ind, col_ind, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_weights, weights, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_dist, dist, nv*sizeof(int2), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_queue, srcArr, srcNeigh * sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_percentage, percentage, sizeof(float), hipMemcpyHostToDevice));


	int size = srcNeigh;
	int *nextSize = (int*)malloc(sizeof(int));
	*nextSize = 0;
	

	int round = 1;
	int temp = 0;
	

	// no approximation. Both signals are negative
	while((size > 0) && (round < nv) && temp < ne && !signal_reduce_execution && !signal_partial_graph_process) { temp += size;

		//printf("NO APPR\n");

		cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));

		cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));

		relaxAtom<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_row_ptr, d_col_ind, d_weights, d_queue, d_nextQueue, size, d_nextSize, d_dist, d_iter); 

		cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));

		(*iter) ++;


		printf("size: %i\n", size);

		size = *nextSize;
		*nextSize = 0;
		std::swap(d_queue, d_nextQueue); // swap frontiers

		//printf("round: %i\n", round);

		(*appr_vals)[2] = round;
		round++;

	}

	//printf("total size: %i\n", temp);

	// Copy outputs to host
	cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));
	
	hipProfilerStop();

	cudaCheck(hipEventRecord(stop, 0));
	cudaCheck(hipEventSynchronize(stop));
	float elapsed;
	cudaCheck(hipEventElapsedTime(&elapsed, start, stop));


	// check for negative cycles

	int neg_cycle = false;

	for (int v2 = 0; v2 < nv; v2++)
	{
		for (int e = (row_ptr)[v2]; e < (row_ptr)[v2+1]; e++)
		{
			int tempDistance = dist[v2].x + (weights)[e];
			int neigh = (col_ind)[e];

			if (tempDistance < dist[neigh].x && dist[v2].x != INT_MAX)
			{
				printf("GPU belman ford Error: negative cycle exists\n");
				neg_cycle = true;
				break;
			}

		}
		if(neg_cycle)
			break;
	}
	printf("NEGATIVE CYCLE CHECK: %i\n", neg_cycle);

	*distance = (int*)malloc((nv+1)*sizeof(int)); 

	for (int i = 0; i < nv; i++)
	{
		(*distance)[i] = dist[i].x;
	}


	// Deallocation
	cudaCheck(hipFree(d_row_ptr));
	cudaCheck(hipFree(d_col_ind));
	cudaCheck(hipFree(d_weights));
	cudaCheck(hipFree(d_dist));


	printf("GPU SBF time(ms): %f\n", elapsed);
	
}