#include "hip/hip_runtime.h"

extern "C" {
#include "dijkstra.cuh"
}

#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <iostream>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


#define cudaCheck(error) \
	if (error != hipSuccess) { \
    	printf("Fatal error: %s at %s:%d\n", \
    		hipGetErrorString(error), \
    		__FILE__, __LINE__); \
    	exit(1); \
 }


// This is Algorithm 4 in the paper

__global__  void cudaSSSPKernel1 ( int *row_ptr, int *col_ind, int *weights,
                                   int *visited, int *distance, unsigned int *temp_distance,
                                   int nv, int ne )
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if ( visited[tid] != 0 )
    {
    	//printf("inside kernel1: %i\n", tid);
        visited[tid] = 0;

        for(int edge = row_ptr[tid]; edge < row_ptr[tid + 1]; edge++)
        {
            int nid = col_ind[edge];

            int w = weights[edge];
            int du = distance[tid];
            int newDist = du + w;

            //if (temp_distance[nid] > (distance[tid] + weights[edge]))
            if ((temp_distance[nid] > newDist) && (du != INT_MAX))
            {
                //temp_distance[nid] = (distance[tid] + weights[edge]);
                //printf("temp_distance[%i]: %i\n", nid, temp_distance[nid]);
            	atomicMin(&temp_distance[nid], newDist);
            }
        }
    }
}


__global__  void cudaSSSPKernel1AtomicMinBlock ( int *row_ptr, int *col_ind, int *weights,
                                   int *visited, int *distance, unsigned int *temp_distance,
                                   int nv, int ne )
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if ( visited[tid] != 0 )
    {
    	//printf("inside kernel1: %i\n", tid);
        visited[tid] = 0;

        for(int edge = row_ptr[tid]; edge < row_ptr[tid + 1]; edge++)
        {
            int nid = col_ind[edge];

            int w = weights[edge];
            int du = distance[tid];
            int newDist = du + w;

            //if (temp_distance[nid] > (distance[tid] + weights[edge]))
            if ((temp_distance[nid] > newDist) && (du != INT_MAX))
            {
                //temp_distance[nid] = (distance[tid] + weights[edge]);
                //printf("temp_distance[%i]: %i\n", nid, temp_distance[nid]);
            	atomicMin_block(&temp_distance[nid], newDist);
            }
        }
    }
}



// This is Algorithm 5 in the paper.  

__global__  void cudaSSSPKernel2 ( int *row_ptr, int *col_ind, int *weights,
                                   int *visited, int *distance, unsigned int *temp_distance)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (distance[tid] > temp_distance[tid])
    {
        distance[tid] = temp_distance[tid];
        visited[tid] = 1;
        //printf("kernel2 -> distance[%i]: %i\n", tid, distance[tid]);
    }

    temp_distance[tid] = distance[tid];
}



__global__  void cudaSSSPKernel1MoreEdges ( int *row_ptr, int *col_ind, int *weights,
                                    int *visited, int *distance, unsigned int *temp_distance,
                                    int nv, int ne, int *min_edges )
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if ( visited[tid] != 0 )
    {
    	//printf("inside kernel1: %i\n", tid);
        visited[tid] = 0;

        if ((row_ptr[tid+1] - row_ptr[tid]) > *min_edges) // process only the nodes which has more than ... edges 
        {
	        for(int edge = row_ptr[tid]; edge < row_ptr[tid + 1]; edge++)
	        {
	            int nid = col_ind[edge];

	            int w = weights[edge];
	            int du = distance[tid];
	            int newDist = du + w;

	            //if (temp_distance[nid] > (distance[tid] + weights[edge]))
	            if ((temp_distance[nid] > newDist) && (du != INT_MAX))
	            {
	                //temp_distance[nid] = (distance[tid] + weights[edge]);
	                //printf("temp_distance[%i]: %i\n", nid, temp_distance[nid]);
	            	atomicMin(&temp_distance[nid], newDist);
	            }
	        }
	    }
    }
}

__global__ void cudaInitDistance(int *distance, unsigned int *temp_distance, int *visited, 
	                             int *nv, int *max_distance)
{
	int bdim = blockDim.x, gdim = gridDim.x, bid = blockIdx.x, tid = threadIdx.x;
	int i = bdim * bid + tid;
	int skip = bdim * gdim;

	for (int k = i; k < *nv; k += skip) 
	{
		distance[k] = *max_distance; //instead of INT_MAX, calculated possible max_distance
		temp_distance[k] = *max_distance;
		visited[k] = 0;
	}

	__syncthreads();

}

// Tells if the algorithm needs to continue running or not.

bool visitEmpty(int *visited, int count)
{
    for(int i = 0; i < count; i++ )
    {
        if (visited[i] == 1)

            return false;
    }

    return true;
}



extern "C"


void apprsdj(const int *row_ptr, const int *col_ind, const int *weights, int **distance, 
	         const int nv, const int ne, int source, int *signals, float **signal_variables, 
	         int max_distance, float *time)
{
	// Initialize GPU variables
	int *d_row_ptr, *d_col_ind, *d_weights, *d_distance, *d_visited, *d_nv, *d_ne,
	    *d_max_distance, *d_min_edges;
	unsigned int *d_temp_distance;

	
	// Initialize CPU variables
	*distance = (int*)malloc(nv*sizeof(int)); 
	int *visited = (int*)calloc(nv, sizeof(int));
	unsigned int *temp_distance = (unsigned int*)malloc(nv*sizeof(unsigned int));


	//SIGNALSs
	int signal_partial_graph_process = signals[4];
	int signal_reduce_execution = signals[3];
	int iter_num = (*signal_variables)[1];
	float *percentage = (float*) malloc (sizeof(float));
	*percentage = (*signal_variables)[2];
	int min_edges = (*signal_variables)[0];
	int signal_atomicMinBlock = signals[5];


	// Allocate device
	cudaCheck(hipMalloc((void **)&d_row_ptr, (nv+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_col_ind, (ne+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_weights, (ne+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_distance, nv*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_visited, nv*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_nv, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_ne, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_temp_distance, nv*sizeof(unsigned int)));
	cudaCheck(hipMalloc((void **)&d_max_distance, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_min_edges, sizeof(int)));

	//Copy inputs to device
	cudaCheck(hipMemcpy(d_row_ptr, row_ptr, (nv+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_col_ind, col_ind, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_weights, weights, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_nv, &nv, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_ne, &ne, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_max_distance, &max_distance, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_min_edges, &min_edges, sizeof(int), hipMemcpyHostToDevice));


	int threadnum = 128;
	int gridSize = (nv + threadnum - 1) / threadnum;  


	cudaInitDistance<<<gridSize, threadnum>>>(d_distance, d_temp_distance, d_visited, d_nv, d_max_distance);

	cudaCheck(hipMemcpy(*distance, d_distance, nv*sizeof(int), hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(temp_distance, d_temp_distance, nv*sizeof(unsigned int), hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(visited, d_visited, nv*sizeof(int), hipMemcpyDeviceToHost));

	(*distance)[source] = 0;
	visited[source] = 1;

	cudaCheck(hipMemcpy(d_distance, *distance, nv*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_visited, visited, nv*sizeof(int), hipMemcpyHostToDevice));

	int count = 0;

	float elapsed;

	if (!signal_reduce_execution && !min_edges && !signal_atomicMinBlock)
	{
		hipEvent_t start;
		hipEvent_t stop;
		cudaCheck(hipEventCreate(&start));
		cudaCheck(hipEventCreate(&stop));
		cudaCheck(hipEventRecord(start, 0));

		// no approximation. Both signals are negative
	    while(!visitEmpty(visited, nv))
	    {
	        // execute the kernel
	        cudaSSSPKernel1<<< gridSize, threadnum >>>( d_row_ptr, d_col_ind, d_weights,
	                                                d_visited, d_distance, d_temp_distance,
	                                                nv, ne );
	       
	        cudaSSSPKernel2<<< gridSize, threadnum >>>( d_row_ptr, d_col_ind, d_weights,
	                                                d_visited, d_distance, d_temp_distance );
	        
	        cudaCheck(hipMemcpy( visited, d_visited, sizeof(int) * nv, hipMemcpyDeviceToHost ));

	        count++;
	        (*signal_variables)[1] = count;
	        
	    }

		cudaCheck(hipEventRecord(stop, 0));
		cudaCheck(hipEventSynchronize(stop));
		cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

		//Copy outputs to host
		cudaCheck(hipMemcpy(*distance, d_distance, nv*sizeof(int), hipMemcpyDeviceToHost));
	}

	if (min_edges && !signal_reduce_execution && !signal_atomicMinBlock)
	{
		hipEvent_t start;
		hipEvent_t stop;
		cudaCheck(hipEventCreate(&start));
		cudaCheck(hipEventCreate(&stop));
		cudaCheck(hipEventRecord(start, 0));

		//min edges to process signal is active. only some edges are processed in kernel
	    while(!visitEmpty(visited, nv))
	    {
	        // execute the kernel
	        cudaSSSPKernel1MoreEdges<<< gridSize, threadnum >>> ( d_row_ptr, d_col_ind, d_weights,
	                                   							  d_visited, d_distance, d_temp_distance,
	                                   							  nv, ne, d_min_edges );

	        cudaSSSPKernel2<<< gridSize, threadnum >>>( d_row_ptr, d_col_ind, d_weights,
	                                                d_visited, d_distance, d_temp_distance );
	        

	        cudaCheck(hipMemcpy( visited, d_visited, sizeof(int) * nv, hipMemcpyDeviceToHost ));
	    }

		cudaCheck(hipEventRecord(stop, 0));
		cudaCheck(hipEventSynchronize(stop));
		cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

		//Copy outputs to host
		cudaCheck(hipMemcpy(*distance, d_distance, nv*sizeof(int), hipMemcpyDeviceToHost));
	}

    
	if (signal_reduce_execution && !min_edges && !signal_atomicMinBlock)
	{
		hipEvent_t start;
		hipEvent_t stop;
		cudaCheck(hipEventCreate(&start));
		cudaCheck(hipEventCreate(&stop));
		cudaCheck(hipEventRecord(start, 0));

		while(!visitEmpty(visited, nv) && count <= iter_num)
	    {
	        // execute the kernel
	        cudaSSSPKernel1<<< gridSize, threadnum >>>( d_row_ptr, d_col_ind, d_weights,
	                                                d_visited, d_distance, d_temp_distance,
	                                                nv, ne );
	       
	        cudaSSSPKernel2<<< gridSize, threadnum >>>( d_row_ptr, d_col_ind, d_weights,
	                                                d_visited, d_distance, d_temp_distance );
	        

	        cudaCheck(hipMemcpy( visited, d_visited, sizeof(int) * nv, hipMemcpyDeviceToHost ));

	        count++;
	    }

		cudaCheck(hipEventRecord(stop, 0));
		cudaCheck(hipEventSynchronize(stop));
		cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

		//Copy outputs to host
		cudaCheck(hipMemcpy(*distance, d_distance, nv*sizeof(int), hipMemcpyDeviceToHost));
	}

	if (min_edges && signal_reduce_execution && !signal_atomicMinBlock)
	{
		hipEvent_t start;
		hipEvent_t stop;
		cudaCheck(hipEventCreate(&start));
		cudaCheck(hipEventCreate(&stop));
		cudaCheck(hipEventRecord(start, 0));

		
	    while(!visitEmpty(visited, nv) && count <= iter_num)
	    {
	        // execute the kernel
	        cudaSSSPKernel1MoreEdges<<< gridSize, threadnum >>> ( d_row_ptr, d_col_ind, d_weights,
	                                   							  d_visited, d_distance, d_temp_distance,
	                                   							  nv, ne, d_min_edges );

	        cudaSSSPKernel2<<< gridSize, threadnum >>>( d_row_ptr, d_col_ind, d_weights,
	                                                d_visited, d_distance, d_temp_distance );
	        

	        cudaCheck(hipMemcpy( visited, d_visited, sizeof(int) * nv, hipMemcpyDeviceToHost ));

	        count++;
	    }

		cudaCheck(hipEventRecord(stop, 0));
		cudaCheck(hipEventSynchronize(stop));
		cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

		//Copy outputs to host
		cudaCheck(hipMemcpy(*distance, d_distance, nv*sizeof(int), hipMemcpyDeviceToHost));
	}

	
	if (signal_atomicMinBlock)
	{
		hipEvent_t start;
		hipEvent_t stop;
		cudaCheck(hipEventCreate(&start));
		cudaCheck(hipEventCreate(&stop));
		cudaCheck(hipEventRecord(start, 0));

	    while(!visitEmpty(visited, nv))
	    {
	        // execute the kernel
	        cudaSSSPKernel1AtomicMinBlock<<< gridSize, threadnum >>>( d_row_ptr, d_col_ind, d_weights,
	                                                d_visited, d_distance, d_temp_distance,
	                                                nv, ne );
	       
	        cudaSSSPKernel2<<< gridSize, threadnum >>>( d_row_ptr, d_col_ind, d_weights,
	                                                d_visited, d_distance, d_temp_distance );
	        
	        cudaCheck(hipMemcpy( visited, d_visited, sizeof(int) * nv, hipMemcpyDeviceToHost ));
	    }

		cudaCheck(hipEventRecord(stop, 0));
		cudaCheck(hipEventSynchronize(stop));
		float elapsed;
		cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

		//Copy outputs to host
		cudaCheck(hipMemcpy(*distance, d_distance, nv*sizeof(int), hipMemcpyDeviceToHost));
		
	    
	}
	
	
	// Deallocation
	cudaCheck(hipFree(d_row_ptr));
	cudaCheck(hipFree(d_col_ind));
	cudaCheck(hipFree(d_weights));
	cudaCheck(hipFree(d_distance));

	printf("GPU SDJ time (ms): %f\n", elapsed);

	*time = elapsed;
}
