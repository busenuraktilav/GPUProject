#include "hip/hip_runtime.h"

extern "C" {
#include "hybrid.cuh"
//#include "../graphio/graphio.h"
}

#include <stdio.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


#define cudaCheck(error) \
	if (error != hipSuccess) { \
    	printf("Fatal error: %s at %s:%d\n", \
    		hipGetErrorString(error), \
    		__FILE__, __LINE__); \
    	exit(1); \
 }


__global__  void CUDA_KERNEL1( int *row_ptr, int *col_ind, int *weights,
                                    int *visited, int *distance, int *temp_distance,
                                    int nv, int ne )
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if ( visited[tid] != 0 )
    {
        visited[tid] = 0;

        for(int edge = row_ptr[tid]; edge < row_ptr[tid + 1]; edge++)
        {
            int nid = col_ind[edge];

            int w = weights[edge];
            int du = distance[tid];
            int newDist = du + w;

            if ((temp_distance[nid] > newDist) && (du != INT_MAX))
            {
            	atomicExch(&temp_distance[nid], newDist);
            	
            }
        }
    }
}
  

__global__  void CUDA_KERNEL2(  int *row_ptr, int *col_ind, int *weights,
                                     int *visited, int *distance, int *temp_distance, int ne)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (distance[tid] > temp_distance[tid])
    {
        distance[tid] = temp_distance[tid];
        visited[tid] = 1;
    }

    temp_distance[tid] = distance[tid];
}


// Tells if the algorithm needs to continue running or not.

bool visited_empty(int *visited, int count)
{
    for(int i = 0; i < count; i++ )
    {
        if (visited[i] == 1)

            return false;
    }

    return true;
}



extern "C"


void shybrid(const int *row_ptr, const int *col_ind, const int *weights, int **distance, int **previous, const int nv, const int ne, int source, int *count, int neg_edge_count)
{
	// Initialize GPU variables
	int *d_row_ptr, *d_col_ind, *d_weights, *d_distance, *d_previous, *d_visited, *d_nv, *d_ne;
	int *d_temp_distance;

	
	// Initialize CPU variables
	*distance = (int*)malloc(nv*sizeof(int)); 
	*previous = (int*)malloc(nv*sizeof(int));
	int *visited = (int*)calloc(nv, sizeof(int));
	int *temp_distance = (int*)malloc(nv*sizeof(int));

	for (int i = 0; i < nv; i++)
	{
		(*distance)[i] = INT_MAX;
		(*previous)[i] = -1;
		temp_distance[i] = INT_MAX;
		visited[i] = 0;
	}

	(*distance)[source] = 0;
	visited[source] = 1;


	// Allocate device
	cudaCheck(hipMalloc((void **)&d_row_ptr, (nv+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_col_ind, (ne+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_weights, (ne+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_distance, nv*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_previous, nv*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_visited, (nv+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_nv, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_ne, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_temp_distance, nv*sizeof(int)));

	//Copy inputs to device
	cudaCheck(hipMemcpy(d_row_ptr, row_ptr, (nv+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_col_ind, col_ind, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_weights, weights, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_distance, *distance, nv*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_nv, &nv, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_ne, &ne, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_visited, visited, (nv+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_temp_distance, temp_distance, nv*sizeof(int), hipMemcpyHostToDevice));



	hipEvent_t start;
	hipEvent_t stop;

	cudaCheck(hipEventCreate(&start));
	cudaCheck(hipEventCreate(&stop));
	cudaCheck(hipEventRecord(start, 0));

	int threadnum = (nv > 1024) ? 1024 : nv;

	int k = neg_edge_count;

	*count = 0;

	int gridSize = (nv + 1023) / 1024; 

	printf("neg_edge_count: %i\n", neg_edge_count);


	while((*count) != (k+2))
    {
        // execute the kernel
        CUDA_KERNEL1<<< gridSize, threadnum >>>( d_row_ptr, d_col_ind, d_weights,
                                                d_visited, d_distance, d_temp_distance,
                                                nv, ne );
       

        CUDA_KERNEL2<<< gridSize, threadnum >>>( d_row_ptr, d_col_ind, d_weights,
                                                d_visited, d_distance, d_temp_distance, ne );
        

        cudaCheck(hipMemcpy( visited, d_visited, sizeof(int) * (nv+1), hipMemcpyDeviceToHost ));
        (*count)++;
    
    }



	printf("count: %i\n", *count);
	
	cudaCheck(hipEventRecord(stop, 0));
	cudaCheck(hipEventSynchronize(stop));
	float elapsed;
	cudaCheck(hipEventElapsedTime(&elapsed, start, stop));
	
	//Copy outputs to host
	cudaCheck(hipMemcpy(*distance, d_distance, nv*sizeof(int), hipMemcpyDeviceToHost));


	// Deallocation
	cudaCheck(hipFree(d_row_ptr));
	cudaCheck(hipFree(d_col_ind));
	cudaCheck(hipFree(d_weights));
	cudaCheck(hipFree(d_distance));
	cudaCheck(hipFree(d_previous));

	printf("GPU SHYBRID time: %f\n", elapsed);

	
}