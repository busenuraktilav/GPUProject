#include "hip/hip_runtime.h"

extern "C" {
#include "bellman_ford.cuh"
}

#include "../../../include/utils/bellman_ford_kernel.cuh"

extern "C"

float apprbf(const int *row_ptr, const int *col_ind, const int *weights, int **distance, 
	        const int nv, const int ne, int source, int iter_num)
{
	int max_distance = INT_MAX;

	// Initialize GPU variables
	int *d_row_ptr, *d_col_ind, *d_weights, *d_nv, *d_max_distance;
	int *d_queue, *d_nextQueue, *d_nextSize, *d_iter;
	int2* d_dist;
	

	int2* dist = (int2*)malloc(nv*sizeof(int2));

	cudaCheck(hipMalloc((void **)&d_row_ptr, (nv+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_col_ind, (ne+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_weights, (ne+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_dist, nv*sizeof(int2)));
	cudaCheck(hipMalloc((void **)&d_nv, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_max_distance, sizeof(int)));

	cudaCheck(hipMemcpy(d_nv, &nv, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_max_distance, &max_distance, sizeof(int), hipMemcpyHostToDevice));


	initialize<<<(nv + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_dist, d_nv, d_max_distance);

	cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));


	float elapsed = 0.0, elapsed0 = 0.0;

	hipEvent_t start0;
	hipEvent_t stop0;
	cudaCheck(hipEventCreate(&start0));
	cudaCheck(hipEventCreate(&stop0));
	cudaCheck(hipEventRecord(start0, 0));

	//To increase parallelism, first process the source vertex
	int srcNeigh = row_ptr[source + 1] - row_ptr[source];
	int *srcArr = (int*)calloc(srcNeigh, sizeof(int));
	int index = 0;
	dist[source].x = 0;

	for (int i = row_ptr[source]; i < row_ptr[source + 1]; i++) { // for each neighbor of source node
		
		int w = weights[i];
		int du = dist[source].x;
		int dv = dist[col_ind[i]].x;

		if (du + w < dv && du != INT_MAX)
		{
			dist[col_ind[i]].x = du + w;
			dist[col_ind[i]].y = 1;
			srcArr[index++] = col_ind[i]; // add to frontier
		}
	}

	cudaCheck(hipEventRecord(stop0, 0));
	cudaCheck(hipEventSynchronize(stop0));
	cudaCheck(hipEventElapsedTime(&elapsed0, start0, stop0));


	int *iter = (int*)malloc(sizeof(int));
	*iter = 2;

	// Allocate device
	
	cudaCheck(hipMalloc((void **)&d_nextSize, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_queue, (nv+1)*sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_nextQueue, (nv+1)*sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_iter, sizeof(int)));
    
	//Copy inputs to device
	cudaCheck(hipMemcpy(d_row_ptr, row_ptr, (nv+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_col_ind, col_ind, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_weights, weights, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_dist, dist, nv*sizeof(int2), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_queue, srcArr, srcNeigh * sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));

	int size = srcNeigh;
	int *nextSize = (int*)malloc(sizeof(int));
	*nextSize = 0;

	int round = 1;

	cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));


	hipEvent_t start;
	hipEvent_t stop;
	cudaCheck(hipEventCreate(&start));
	cudaCheck(hipEventCreate(&stop));
	cudaCheck(hipEventRecord(start, 0));


	//hipProfilerStart();

	int *d_size;
	cudaCheck(hipMalloc((void **)&d_size, sizeof(int)));

	while( (round < iter_num+1) ) {

		cudaCheck(hipMemcpy(d_size, &size, sizeof(int), hipMemcpyHostToDevice));
		cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
		cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));

		relaxAtomicOperations<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_row_ptr, d_col_ind, d_weights, d_queue, d_nextQueue, d_size, d_nextSize, d_dist, d_iter);

		cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));

		(*iter) ++;
		printf("size: %i\n", size);
		size = *nextSize;
		*nextSize = 0;
		std::swap(d_queue, d_nextQueue); // swap frontiers
		round++;

	}

	//hipProfilerStop();

	cudaCheck(hipEventRecord(stop, 0));
	cudaCheck(hipEventSynchronize(stop));
	cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

	// Copy outputs to host
	cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));


	// check for negative cycles
	int neg_cycle = false;

	for (int v2 = 0; v2 < nv; v2++)
	{
		for (int e = (row_ptr)[v2]; e < (row_ptr)[v2+1]; e++)
		{
			int tempDistance = dist[v2].x + (weights)[e];
			int neigh = (col_ind)[e];

			if ((tempDistance < dist[neigh].x) && (dist[v2].x != INT_MAX))
			{
				printf("GPU belman ford Error: negative cycle exists\n");
				neg_cycle = true;
				break;
			}

		}
		if(neg_cycle)
			break;
	}

	printf("NEGATIVE CYCLE CHECK: %i\n", neg_cycle);


	*distance = (int*)malloc((nv+1)*sizeof(int)); 

	for (int i = 0; i < nv; i++)
	{
		(*distance)[i] = dist[i].x;
	}


	// Deallocation
	cudaCheck(hipFree(d_row_ptr));
	cudaCheck(hipFree(d_col_ind));
	cudaCheck(hipFree(d_weights));
	cudaCheck(hipFree(d_dist));
	cudaCheck(hipFree(d_nv));
	cudaCheck(hipFree(d_max_distance));
	cudaCheck(hipFree(d_queue));
	cudaCheck(hipFree(d_nextQueue));
	cudaCheck(hipFree(d_nextSize));
	cudaCheck(hipFree(d_iter));
	
	printf("GPU SBF time(ms): %f\n", elapsed+elapsed0);

	return (elapsed+elapsed0);

}