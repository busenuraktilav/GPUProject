#include "hip/hip_runtime.h"

extern "C" {
#include "bellman_ford.cuh"
}

#include "../../../include/utils/bellman_ford_kernel.cuh"

extern "C"

float apprbf(const int *row_ptr, const int *col_ind, const int *weights, int **distance, 
	        const int nv, const int ne, int source, int min_edges, int max_edges)
{
	int max_distance = INT_MAX;

	// Initialize GPU variables
	int *d_row_ptr, *d_col_ind, *d_weights, *d_nv, *d_max_distance;
	int *d_queue, *d_nextQueue, *d_nextSize, *d_iter, *d_min_edges, *d_max_edges;
	int2* d_dist;
	

	int2* dist = (int2*)malloc(nv*sizeof(int2));

	cudaCheck(hipMalloc((void **)&d_row_ptr, (nv+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_col_ind, (ne+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_weights, (ne+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_dist, nv*sizeof(int2)));
	cudaCheck(hipMalloc((void **)&d_nv, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_max_distance, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_min_edges, sizeof(int))); //for kernel cudarelaxAtomicMoreEdges, min edges
	cudaCheck(hipMalloc((void **)&d_max_edges, sizeof(int)));

	cudaCheck(hipMemcpy(d_nv, &nv, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_max_distance, &max_distance, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_min_edges, &min_edges, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_max_edges, &max_edges, sizeof(int), hipMemcpyHostToDevice));

	initialize<<<(nv + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_dist, d_nv, d_max_distance);

	cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));

	float elapsed = 0.0, elapsed0 = 0.0;

	hipEvent_t start0;
	hipEvent_t stop0;
	cudaCheck(hipEventCreate(&start0));
	cudaCheck(hipEventCreate(&stop0));
	cudaCheck(hipEventRecord(start0, 0));

	//To increase parallelism, first process the source vertex
	int srcNeigh = row_ptr[source + 1] - row_ptr[source];
	int *srcArr = (int*)calloc(srcNeigh, sizeof(int));
	int index = 0;
	dist[source].x = 0;

	for (int i = row_ptr[source]; i < row_ptr[source + 1]; i++) { // for each neighbor of source node
		
		int w = weights[i];
		int du = dist[source].x;
		int dv = dist[col_ind[i]].x;

		if (du + w < dv && du != INT_MAX)
		{
			dist[col_ind[i]].x = du + w;
			dist[col_ind[i]].y = 1;
			srcArr[index++] = col_ind[i]; // add to frontier
		}
	}

	cudaCheck(hipEventRecord(stop0, 0));
	cudaCheck(hipEventSynchronize(stop0));
	cudaCheck(hipEventElapsedTime(&elapsed0, start0, stop0));


	int *iter = (int*)malloc(sizeof(int));
	*iter = 2;

	// Allocate device
	
	cudaCheck(hipMalloc((void **)&d_nextSize, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_queue, (nv+1)*sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_nextQueue, (nv+1)*sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_iter, sizeof(int)));
    
	//Copy inputs to device
	cudaCheck(hipMemcpy(d_row_ptr, row_ptr, (nv+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_col_ind, col_ind, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_weights, weights, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_dist, dist, nv*sizeof(int2), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_queue, srcArr, srcNeigh * sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));

	int size = srcNeigh;
	int *nextSize = (int*)malloc(sizeof(int));
	*nextSize = 0;

	cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));

	//min edges to process signal is active. only some edges are processed in kernel

	hipEvent_t start;
	hipEvent_t stop;
	cudaCheck(hipEventCreate(&start));
	cudaCheck(hipEventCreate(&stop));
	cudaCheck(hipEventRecord(start, 0));

	//hipProfilerStart();

	int *d_size;
	cudaCheck(hipMalloc((void **)&d_size, sizeof(int)));

	while((size > 0)) { 

		cudaCheck(hipMemcpy(d_size, &size, sizeof(int), hipMemcpyHostToDevice));
		cudaCheck(hipMemcpy(d_iter, iter, sizeof(int), hipMemcpyHostToDevice));
		cudaCheck(hipMemcpy(d_nextSize, nextSize, sizeof(int), hipMemcpyHostToDevice));

		cudarelaxAtomicMoreLessEdgesBlock<<<(size + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>
		                           (d_row_ptr, d_col_ind, d_weights, d_queue, d_nextQueue, d_size, d_nextSize, d_dist, 
		                           	d_iter, d_min_edges, d_max_edges); 

		cudaCheck(hipMemcpy(nextSize, d_nextSize, sizeof(int), hipMemcpyDeviceToHost));

		(*iter) ++;
		printf("size: %i\n", size);
		size = *nextSize;
		*nextSize = 0;
		std::swap(d_queue, d_nextQueue); // swap frontiers
	}

	//hipProfilerStop();

	cudaCheck(hipEventRecord(stop, 0));
	cudaCheck(hipEventSynchronize(stop));
	cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

	// Copy outputs to host
	cudaCheck(hipMemcpy(dist, d_dist, nv*sizeof(int2), hipMemcpyDeviceToHost));

	// check for negative cycles
	int neg_cycle = false;

	for (int v2 = 0; v2 < nv; v2++)
	{
		for (int e = (row_ptr)[v2]; e < (row_ptr)[v2+1]; e++)
		{
			int tempDistance = dist[v2].x + (weights)[e];
			int neigh = (col_ind)[e];

			if ((tempDistance < dist[neigh].x) && (dist[v2].x != INT_MAX))
			{
				printf("GPU belman ford Error: negative cycle exists\n");
				neg_cycle = true;
				break;
			}

		}
		if(neg_cycle)
			break;
	}

	printf("NEGATIVE CYCLE CHECK: %i\n", neg_cycle);


	*distance = (int*)malloc((nv+1)*sizeof(int)); 

	for (int i = 0; i < nv; i++)
	{
		(*distance)[i] = dist[i].x;
	}


	// Deallocation
	cudaCheck(hipFree(d_row_ptr));
	cudaCheck(hipFree(d_col_ind));
	cudaCheck(hipFree(d_weights));
	cudaCheck(hipFree(d_dist));
	cudaCheck(hipFree(d_nv));
	cudaCheck(hipFree(d_max_distance));
	cudaCheck(hipFree(d_queue));
	cudaCheck(hipFree(d_nextQueue));
	cudaCheck(hipFree(d_nextSize));
	cudaCheck(hipFree(d_iter));
	cudaCheck(hipFree(d_min_edges));
	cudaCheck(hipFree(d_max_edges));


	printf("GPU SBF time(ms): %f\n", elapsed+elapsed0);

	return (elapsed+elapsed0);
	
}