
extern "C" {
#include "hybrid.cuh"
}

#include "../../../include/utils/dijkstra_kernel.cuh"

extern "C"


float apprshybrid(const int *row_ptr, const int *col_ind, const int *weights, int **distance, 
	        const int nv, const int ne, int source, int iter_num, int neg_edge_count)
{
	// Initialize GPU variables
	int *d_row_ptr, *d_col_ind, *d_weights, *d_distance, *d_visited, *d_nv, *d_ne,
	    *d_max_distance;
	unsigned int *d_temp_distance;

	
	// Initialize CPU variables
	int max_distance = INT_MAX;
	*distance = (int*)malloc(nv*sizeof(int)); 
	int *visited = (int*)calloc(nv, sizeof(int));
	unsigned int *temp_distance = (unsigned int*)malloc(nv*sizeof(unsigned int));


	// Allocate device
	cudaCheck(hipMalloc((void **)&d_row_ptr, (nv+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_col_ind, (ne+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_weights, (ne+1)*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_distance, nv*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_visited, nv*sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_nv, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_ne, sizeof(int)));
	cudaCheck(hipMalloc((void **)&d_temp_distance, nv*sizeof(unsigned int)));
	cudaCheck(hipMalloc((void **)&d_max_distance, sizeof(int)));

	//Copy inputs to device
	cudaCheck(hipMemcpy(d_row_ptr, row_ptr, (nv+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_col_ind, col_ind, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_weights, weights, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_nv, &nv, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_ne, &ne, sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_max_distance, &max_distance, sizeof(int), hipMemcpyHostToDevice));

	initialize<<<(nv + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK>>>(d_distance, d_temp_distance, d_visited, d_nv, d_max_distance);

	cudaCheck(hipMemcpy(*distance, d_distance, nv*sizeof(int), hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(temp_distance, d_temp_distance, nv*sizeof(unsigned int), hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(visited, d_visited, nv*sizeof(int), hipMemcpyDeviceToHost));

	(*distance)[source] = 0;
	temp_distance[source] = 0;
	visited[source] = 1;

	cudaCheck(hipMemcpy(d_distance, *distance, nv*sizeof(int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_temp_distance, temp_distance, nv*sizeof(unsigned int), hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(d_visited, visited, nv*sizeof(int), hipMemcpyHostToDevice));

	int k = neg_edge_count;
	int r = 0;
	printf("neg_edge_count: %i\n", neg_edge_count);

	float elapsed;
	hipEvent_t start;
	hipEvent_t stop;
	cudaCheck(hipEventCreate(&start));
	cudaCheck(hipEventCreate(&stop));
	cudaCheck(hipEventRecord(start, 0));
        

	while(r != (k+2))
  {
      for (int i = 0; i < nv; i++)
		
				visited[i] = 0;

		visited[source] = 1;
		cudaCheck(hipMemcpy(d_visited, visited, (nv)*sizeof(int), hipMemcpyHostToDevice));


		int count = 0;

		while(!visitedEmpty(visited, nv) && count < iter_num)
	  {
	        // execute the kernel
	        
	        CUDA_SSSP_KERNEL1_Block<<< (nv + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK >>>
	                           ( d_row_ptr, d_col_ind, d_weights, d_visited, d_distance, d_temp_distance,
	                             nv, ne );
	       
	        CUDA_SSSP_KERNEL2<<< (nv + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK >>>
	                           ( d_row_ptr, d_col_ind, d_weights, d_visited, d_distance, d_temp_distance );
	        

	        cudaCheck(hipMemcpy( visited, d_visited, sizeof(int) * nv, hipMemcpyDeviceToHost ));

	        count++;
	  }
	  r++;

	}

	cudaCheck(hipEventRecord(stop, 0));
	cudaCheck(hipEventSynchronize(stop));
	cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

	//Copy outputs to host
	cudaCheck(hipMemcpy(*distance, d_distance, nv*sizeof(int), hipMemcpyDeviceToHost));


	// Deallocation
	cudaCheck(hipFree(d_row_ptr));
	cudaCheck(hipFree(d_col_ind));
	cudaCheck(hipFree(d_weights));
	cudaCheck(hipFree(d_distance));

	printf("GPU SHYBRID time (ms): %f\n", elapsed);

	return elapsed;
}
