extern "C" {
#include "dijkstra.cuh"
}

#include "../../../include/utils/dijkstra_kernel.cuh"

extern "C"

float apprsdj(const int *row_ptr, const int *col_ind, const int *weights, int **distance, 
	     const int nv, const int ne, int source)
{
    // Initialize GPU variables
    int *d_row_ptr, *d_col_ind, *d_weights, *d_distance, *d_visited, *d_nv, *d_ne;
    unsigned int *d_temp_distance;


    // Initialize CPU variables
    *distance = (int*)malloc(nv*sizeof(int)); 
    int *visited = (int*)calloc(nv, sizeof(int));
    unsigned int *temp_distance = (unsigned int*)malloc(nv*sizeof(unsigned int));

    for (int i = 0; i < nv; i++)
    {
	(*distance)[i] = INT_MAX;
	temp_distance[i] = INT_MAX;
	visited[i] = 0;
    }

    //printf("source: %i\n", source);

    (*distance)[source] = 0;
    temp_distance[source] = 0;
    visited[source] = 1;


    // Allocate device
    cudaCheck(hipMalloc((void **)&d_row_ptr, (nv+1)*sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_col_ind, (ne+1)*sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_weights, (ne+1)*sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_distance, nv*sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_visited, nv*sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_nv, sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_ne, sizeof(int)));
    cudaCheck(hipMalloc((void **)&d_temp_distance, nv*sizeof(unsigned int)));

    //Copy inputs to device
    cudaCheck(hipMemcpy(d_row_ptr, row_ptr, (nv+1)*sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_col_ind, col_ind, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_weights, weights, (ne+1)*sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_distance, *distance, nv*sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_nv, &nv, sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_ne, &ne, sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_visited, visited, nv*sizeof(int), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_temp_distance, temp_distance, nv*sizeof(unsigned int), hipMemcpyHostToDevice));


    float elapsed;
    hipEvent_t start;
    hipEvent_t stop;
    cudaCheck(hipEventCreate(&start));
    cudaCheck(hipEventCreate(&stop));
    cudaCheck(hipEventRecord(start, 0));

    while(!visitedEmpty(visited, nv))
    { 
        // execute the kernel
        CUDA_SSSP_KERNEL1_Block<<< (nv + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK >>>( d_row_ptr, d_col_ind, d_weights,
                                                d_visited, d_distance, d_temp_distance,
                                                nv, ne );
       

        CUDA_SSSP_KERNEL2<<< (nv + N_THREADS_PER_BLOCK - 1) / N_THREADS_PER_BLOCK, N_THREADS_PER_BLOCK >>>( d_row_ptr, d_col_ind, d_weights,
                                                d_visited, d_distance, d_temp_distance );
        

        cudaCheck(hipMemcpy( visited, d_visited, sizeof(int) * nv, hipMemcpyDeviceToHost ));
        
    }
	
    cudaCheck(hipEventRecord(stop, 0));
    cudaCheck(hipEventSynchronize(stop));
    cudaCheck(hipEventElapsedTime(&elapsed, start, stop));

    //Copy outputs to host
    cudaCheck(hipMemcpy(*distance, d_distance, nv*sizeof(int), hipMemcpyDeviceToHost));


    // Deallocation
    cudaCheck(hipFree(d_row_ptr));
    cudaCheck(hipFree(d_col_ind));
    cudaCheck(hipFree(d_weights));
    cudaCheck(hipFree(d_distance));

    printf("GPU SDJ time(ms): %f\n", elapsed);

    return elapsed;
}
